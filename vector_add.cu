#include "hip/hip_runtime.h"
// -*- c -*-

__global__ void add_vectors_kernel(float *A, float *B, float *C, int N) {
    // Determine which element this thread is computing
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;
    
    // Compute a single element of the result vector (if the element is valid)
    if (thread_id < N) C[thread_id] = A[thread_id] + B[thread_id];
}

// Returns the vector sum A + B (computed on the GPU)
float *GPU_add_vectors(float *A_CPU, float *B_CPU, int N) {
    
    // Allocate GPU memory for the inputs and the result
    int vector_size = N * sizeof(float);
    float *A_GPU, *B_GPU, *C_GPU;
    hipMalloc((void **) &A_GPU, vector_size);
    hipMalloc((void **) &B_GPU, vector_size);
    hipMalloc((void **) &C_GPU, vector_size);
    
    // Transfer the input vectors to GPU memory
    hipMemcpy(A_GPU, A_CPU, vector_size, hipMemcpyHostToDevice);
    hipMemcpy(B_GPU, B_CPU, vector_size, hipMemcpyHostToDevice);
    
    // Determine the number of thread blocks in the x- and y-dimension
    int num_blocks = (int) ((float) (N + threads_per_block - 1) / (float) threads_per_block);
    int max_blks_per_dim = 65535;
    int num_blocks_y = (int) ((float) (num_blocks + max_blocks_per_dimension - 1) / (float) max_blks_per_dim);
    int num_blocks_x = (int) ((float) (num_blocks + num_blocks_y - 1) / (float) num_blocks_y);
    dim3 grid_size(num_blocks_x, num_blocks_y, 1);
    
    // Execute the kernel to compute the vector sum on the GPU
    add_vectors_kernel <<< grid_size , threads_per_block >>> (A_GPU, B_GPU, C_GPU, N);
    
    // Allocate CPU memory for the result
    float *C_CPU = (float *) malloc(vector_size);
    
    // Transfer the result from the GPU to the CPU
    hipMemcpy(C_CPU, C_GPU, vector_size, hipMemcpyDeviceToHost);
    
    // Free the GPU memory
    hipFree(A_GPU);
    hipFree(B_GPU);
    hipFree(C_GPU);
    
    return C_CPU;
}

